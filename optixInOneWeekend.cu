#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
#include <optix.h>

#include "optixInOneWeekend.h"
#include "random.h"

#include <sutil/vec_math.h>
#include <cuda/helpers.h>

extern "C" {
__constant__ Params params;
}


//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

struct SurfaceInfo
{
    // 発光度
    float3 emission;
    // 物体表面の色
    float3 albedo;
    // 衝突位置
    float3 p;
    // レイの方向
    float3 direction;
    // 法線
    float3 n;
    // テクスチャ座標
    float2 texcoord;

    // 乱数のシード値
    unsigned int seed;
    // 光線追跡を終了するか否か
    int trace_terminate;

    // マテリアル用のデータとCallablesプログラムのID
    Material material;
};

//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

static __forceinline__ __device__ void* unpackPointer( unsigned int i0, unsigned int i1 )
{
    const unsigned long long uptr = static_cast<unsigned long long>( i0 ) << 32 | i1;
    void*           ptr = reinterpret_cast<void*>( uptr );
    return ptr;
}

// ポインタをunsigned long longに変換してから、前側32bitをi0に、後側32bitをi1に格納する
static __forceinline__ __device__ void  packPointer( void* ptr, unsigned int& i0, unsigned int& i1 )
{
    const unsigned long long uptr = reinterpret_cast<unsigned long long>( ptr );
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

// 0番目と1番目のペイロードにパックされているSurfaceInfoのポインタを取得
static __forceinline__ __device__ SurfaceInfo* getSurfaceInfo()
{
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return reinterpret_cast<SurfaceInfo*>( unpackPointer( u0, u1 ) );
}

static __forceinline__ __device__ float3 randomInUnitSphere(unsigned int& seed) {
    while (true)
    {
        float3 v = make_float3(rnd(seed) * 2.0f - 1.0f, rnd(seed) * 2.0f - 1.0f, rnd(seed) * 2.0f - 1.0f);
        if (dot(v, v) >= 1.0f) continue;
        return v;
    }
}

static __forceinline__ __device__ float3 randomSampleHemisphere(unsigned int& seed, const float3& normal)
{
    const float3 vec_in_sphere = randomInUnitSphere(seed);
    if (dot(vec_in_sphere, normal) > 0.0f)
        return vec_in_sphere;
    else
        return -vec_in_sphere;
}

static __forceinline__ __device__ float fresnel(float cosine, float ref_idx)
{
    float r0 = (1 - ref_idx) / (1 + ref_idx);
    r0 = r0 * r0;
    return r0 + (1 - r0) * powf((1 - cosine), 5.0f);
}

static __forceinline__ __device__ float3 refract(const float3& uv, const float3& n, float etai_over_etat) {
    auto cos_theta = fminf(dot(-uv, n), 1.0f);
    float3 r_out_perp = etai_over_etat * (uv + cos_theta * n);
    float3 r_out_parallel = -sqrtf(fabs(1.0f - dot(r_out_perp, r_out_perp))) * n;
    return r_out_perp + r_out_parallel;
}

static __forceinline__ __device__ float3 refract(const float3& wi, const float3& n, float cos_i, float ni, float nt) {
    float nt_ni = nt / ni;
    float ni_nt = ni / nt;
    float D = sqrtf(nt_ni * nt_ni - (1.0f - cos_i * cos_i)) - cos_i;
    return ni_nt * (wi - D * n);
}

static __forceinline__ __device__ void trace(
        OptixTraversableHandle handle,
        float3                 ray_origin,
        float3                 ray_direction,
        float                  tmin,
        float                  tmax,
        SurfaceInfo*           si
        )
{
    // SurfaceInfoのポインタを2つのペイロードにパックする
    unsigned int u0, u1;
    packPointer( si, u0, u1 );
    optixTrace(
            handle,
            ray_origin,
            ray_direction,
            tmin,
            tmax,
            0.0f,                // rayTime
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_NONE,
            0,        // SBT offset
            1,        // SBT stride
            0,        // missSBTIndex
            u0, u1 );
}

//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

extern "C" __global__ void __raygen__pinhole()
{
    const int w = params.width; 
    const int h = params.height;
    const float3 eye = params.eye;
    const float3 U = params.U;
    const float3 V = params.V; 
    const float3 W = params.W;
    const uint3 idx = optixGetLaunchIndex();
    const int subframe_index = params.subframe_index;
    const int samples_per_launch = params.samples_per_launch;

    // 現在のスレッドIDから乱数用のシード値を生成
    unsigned int seed = tea<4>(idx.y * w + idx.x, subframe_index);

    float3 result = make_float3(0.0f);
    for (int i = 0; i < samples_per_launch; i++)
    {
        const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));

        const float2 d = 2.0f * make_float2(
            ((float)idx.x + subpixel_jitter.x) / (float)w, 
            ((float)idx.y + subpixel_jitter.y) / (float)h
        ) - 1.0f;

        // 光線の向きと原点を設定
        float3 ray_direction = normalize(d.x * U + d.y * V + W);
        float3 ray_origin = eye;

        SurfaceInfo si;
        si.emission = make_float3(0.0f);
        si.albedo = make_float3(0.0f);
        si.trace_terminate = false;
        si.seed = seed;

        float3 throughput = make_float3(1.0f);

        int depth = 0;
        for (;;)
        {
            if (depth >= params.max_depth)
                break;

            // IASに対してレイトレース
            trace(params.handle, ray_origin, ray_direction, 0.01f, 1e16f, &si);

            if (si.trace_terminate) {
                result += si.emission * throughput;
                break;
            }

            // Direct callable関数を使って各マテリアルにおける散乱方向とマテリアルの色を計算
            float3 scattered;
            optixDirectCall<void, SurfaceInfo*, void*, float3&>(
                si.material.prg_id, &si, si.material.data, scattered
            );

            throughput *= si.albedo;

            ray_origin = si.p;
            ray_direction = scattered;

            ++depth;
        }
    }

    const unsigned int image_index = idx.y * params.width + idx.x;
    float3 accum_color = result / static_cast<float>(params.samples_per_launch);

    if (subframe_index > 0)
    {
        const float a = 1.0f / static_cast<float>(subframe_index + 1);
        const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    // 取得した輝度値を出力バッファに書き込む
    params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
    params.frame_buffer[image_index] = make_color(accum_color);
}

extern "C" __global__ void __miss__radiance()
{
    const MissData* miss = (MissData*)optixGetSbtDataPointer();

    SurfaceInfo* si = getSurfaceInfo();

    // ベクトルのy成分から背景色を計算
    const float3 unit_direction = normalize(optixGetWorldRayDirection());
    const float t = 0.5f * (unit_direction.y + 1.0f);
    si->emission = (1.0f - t) * make_float3(1.0f) + t * make_float3(0.5f, 0.7f, 1.0f);
    si->trace_terminate      = true;
}

extern "C" __global__ void __closesthit__mesh()
{
    // Shader binding tableからデータを取得
    HitGroupData* data = (HitGroupData*)optixGetSbtDataPointer();
    const MeshData* mesh_data = (MeshData*)data->shape_data;

    const int    prim_idx        = optixGetPrimitiveIndex();
    const float3 direction         = optixGetWorldRayDirection();
    const uint3 index = mesh_data->indices[prim_idx];

    // 三角形の重心座標(u,v)を三角形のテクスチャ座標とする
    const float2 texcoord = optixGetTriangleBarycentrics();

    // メッシュデータから頂点を取得し、法線計算
    const float3 v0   = mesh_data->vertices[ index.x ];
    const float3 v1   = mesh_data->vertices[ index.y ];
    const float3 v2   = mesh_data->vertices[ index.z ];
    const float3 N  = normalize( cross( v1-v0, v2-v0 ) );

    // レイと三角形の交点を計算
    const float3 P    = optixGetWorldRayOrigin() + optixGetRayTmax()*direction;

    // PayloadからSurfaceInfoのポインタを取得し、交点上の情報を格納
    SurfaceInfo* si = getSurfaceInfo();

    // SurfaceInfoに交点における情報を格納する
    si->p = P;
    si->direction = direction;
    si->n = N;
    si->texcoord = texcoord;
    // HitGroupDataに紐付いているマテリアル情報をSurfaceInfoに紐付ける
    si->material = data->material;
}

extern "C" __global__ void __intersection__sphere()
{
    // Shader binding tableからデータを取得
    HitGroupData* data = (HitGroupData*)optixGetSbtDataPointer();
    // AABBとの交差判定が認められた球体のGAS内のIDを取得
    const int prim_idx = optixGetPrimitiveIndex();
    const SphereData sphere_data = ((SphereData*)data->shape_data)[prim_idx];

    const float3 center = sphere_data.center;
    const float radius = sphere_data.radius;

    // オブジェクト空間におけるレイの原点と方向を取得
    const float3 origin = optixGetObjectRayOrigin();
    const float3 direction = optixGetObjectRayDirection();
    // レイの最小距離と最大距離を取得
    const float tmin = optixGetRayTmin();
    const float tmax = optixGetRayTmax();

    // 球体との交差判定処理（判別式を解いて、距離tを計算)
    const float3 oc = origin - center;
    const float a = dot(direction, direction);
    const float half_b = dot(oc, direction);
    const float c = dot(oc, oc) - radius * radius;

    const float discriminant = half_b * half_b - a * c;
    if (discriminant < 0) return;
    
    const float sqrtd = sqrtf(discriminant);

    float root = (-half_b - sqrtd) / a;
    if (root < tmin || tmax < root)
    {
        root = (-half_b + sqrtd) / a;
        if (root < tmin || tmax < root)
            return;
    }

    // オブジェクト空間におけるレイと球の交点を計算
    const float3 P = origin + root * direction;
    const float3 normal = (P - center) / radius;

    // 球体におけるテクスチャ座標を算出 (Z up)と仮定して、xとyから方位角、zから仰角を計算
    float phi = atan2(normal.y, normal.x);
    if (phi < 0) phi += 2.0f * M_PIf;
    const float theta = acosf(normal.z);
    const float2 texcoord = make_float2(phi / (2.0f * M_PIf), theta / M_PIf);

    // レイと球の交差判定を認める
    optixReportIntersection(root, 0, 
        __float_as_int(normal.x), __float_as_int(normal.y), __float_as_int(normal.z),
        __float_as_int(texcoord.x), __float_as_int(texcoord.y)
    );
}

extern "C" __global__ void __closesthit__sphere()
{
    // Shader binding tableからデータを取得
    HitGroupData* data = (HitGroupData*)optixGetSbtDataPointer();

    // 0 - 2番目のAttributeからIntersectionプログラムで計算した法線を取得
    const float3 local_n = make_float3(
        __int_as_float(optixGetAttribute_0()),
        __int_as_float(optixGetAttribute_1()),
        __int_as_float(optixGetAttribute_2())
    );
    // Instanceに紐付いている行列からオブジェクト空間における法線をグローバル空間にマップする
    const float3 world_n = normalize(optixTransformNormalFromObjectToWorldSpace(local_n));

    // 3 - 4番目のAttributeからテクスチャ座標を取得
    const float2 texcoord = make_float2(
        __int_as_float(optixGetAttribute_3()),
        __int_as_float(optixGetAttribute_4())
    );

    // グローバル空間におけるレイの原点と方向を計算し、交点座標の位置を計算
    const float3 origin = optixGetWorldRayOrigin();
    const float3 direction = optixGetWorldRayDirection();
    const float3 P = origin + optixGetRayTmax() * direction;

    // PayloadからSurfaceInfoのポインタを取得し、交点上の情報を格納
    SurfaceInfo* si = getSurfaceInfo();
    si->p = P;
    si->n = world_n;
    si->direction = direction;
    si->texcoord = texcoord;
    // HitGroupDataに紐付いているマテリアル情報をSurfaceInfoに紐付ける
    si->material = data->material;
}

extern "C" __device__ void __direct_callable__lambertian(SurfaceInfo* si, void* material_data, float3& scattered)
{
    const LambertianData* lambertian = (LambertianData*)material_data;

    // Direct callableプログラムによって、テクスチャ色を取得
    const float4 color = optixDirectCall<float4, SurfaceInfo*, void*>(
        lambertian->texture_prg_id, si, lambertian->texture_data
        );
    si->albedo = make_float3(color);

    si->n = faceforward(si->n, -si->direction, si->n);

    unsigned int seed = si->seed;
    float3 wi = randomSampleHemisphere(seed, si->n);
    scattered = normalize(wi);
    si->trace_terminate = false;
    si->emission = make_float3(0.0f);
}

extern "C" __device__ void __direct_callable__dielectric(SurfaceInfo* si, void* material_data, float3& scattered)
{
    const DielectricData* dielectric = (DielectricData*)material_data;
    // Direct callableプログラムによって、テクスチャ色を取得
    const float4 color = optixDirectCall<float4, SurfaceInfo*, void*>(
        dielectric->texture_prg_id, si, dielectric->texture_data
        );

    const float ior = dielectric->ior;
    const float3 in_direction = si->direction;

    si->albedo = make_float3(color);
    float cos_theta = dot(in_direction, si->n);
    bool into = cos_theta < 0;
    const float3 outward_normal = into ? si->n : -si->n;
    const float refraction_ratio = into ? (1.0 / ior) : ior;

    float3 unit_direction = normalize(in_direction);
    cos_theta = fabs(cos_theta);
    float sin_theta = sqrt(1.0 - cos_theta * cos_theta);

    unsigned int seed = si->seed;
    bool cannot_refract = refraction_ratio * sin_theta > 1.0;
    if (cannot_refract || rnd(seed) < fresnel(cos_theta, refraction_ratio))
        scattered = reflect(unit_direction, si->n);
    else
        scattered = refract(unit_direction, outward_normal, refraction_ratio);
    si->trace_terminate = false;
    si->emission = make_float3(0.0f);
    si->seed = seed;
}

extern "C" __device__ void __direct_callable__metal(SurfaceInfo* si, void* material_data, float3& scattered)
{
    const MetalData* metal = (MetalData*)material_data;
    // Direct callableプログラムによって、テクスチャ色を取得
    const float4 color = optixDirectCall<float4, SurfaceInfo*, void*>(
        metal->texture_prg_id, si, metal->texture_data
        );

    unsigned int seed = si->seed;
    scattered = reflect(si->direction, si->n) + metal->fuzz * randomInUnitSphere(seed);
    si->albedo = make_float3(color);
    si->trace_terminate = false;
    si->emission = make_float3(0.0f);
    si->seed = seed;
}

extern "C" __device__ float4 __direct_callable__constant(SurfaceInfo* /* si */ , void* texture_data)
{
    const ConstantData* constant = (ConstantData*)texture_data;
    return constant->color;
}

extern "C" __device__ float4 __direct_callable__checker(SurfaceInfo* si, void* texture_data)
{
    const CheckerData* checker = (CheckerData*)texture_data;
    const bool is_odd = sinf(si->texcoord.x * M_PIf * checker->scale) * sinf(si->texcoord.y * M_PIf * checker->scale) < 0;
    return is_odd ? checker->color1 : checker->color2;
}